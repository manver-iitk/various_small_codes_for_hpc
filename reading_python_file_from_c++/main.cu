#include <iostream>
#include <Python.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <complex.h>

PyObject *pName, *pModule, *pFunc, *pArgs, *pValue;

std::complex<double> data1;
Py_complex data;

__forceinline__ void python_err_check(PyObject *data_pointer, int line, std::string file_name) // Python Error Checker
{
    if (data_pointer == NULL)
    {
        std::cout << "Error in Python Functions call at line no " << line << "\n In File " << file_name << " , aborting " << std::endl;
        exit(0);
    }
}

int main(int argc, char **argv)
{

    // ############ initialize the Python reading #########
    Py_Initialize();

    PyRun_SimpleString("import sys\n"
                       "import os\n"
                       "sys.path.append(os.getcwd())\n");
    // ####################################################

    // Opening Python Script file
    pName = PyUnicode_FromString((char *)"para");
    python_err_check(pName, __LINE__, __FILE__);
    pModule = PyImport_Import(pName);
    python_err_check(pModule, __LINE__, __FILE__);

    // Opening The function named return_data2_type in python script
    pFunc = PyObject_GetAttrString(pModule, (char *)"return_data2_type");
    python_err_check(pFunc, __LINE__, __FILE__);
    pValue = PyObject_CallObject(pFunc, nullptr);
    python_err_check(pValue, __LINE__, __FILE__);

    // reading the complex datatype from the python file 
    data = (PyComplex_AsCComplex(PyList_GET_ITEM(pValue, 0)));
    std::cout << data.imag;
    std::cout << data.real;
    std::cout << std::endl;
    data1 = std::complex<double>(data.real, data.imag);
    std::cout << data1;
    std::cout << std::endl;

    Py_Finalize();
    return 0;
}