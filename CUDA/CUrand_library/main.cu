#include "hip/hip_runtime.h"
#include "header.cuh"


__global__ void init_curand_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(clock64(), id, 0, &state[0]);

}


__global__ void generate_kernel(hiprandState *state,
                                int n,
                                double *result, double *result_2)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int count = 0;
    unsigned int x;

    /* Copy state to local memory for efficiency */
    hiprandState localState;

    hiprand_init(clock64(), id, 0, &localState);

    result[id] = hiprand_uniform_double(&localState);
    result_2[id] = hiprand_uniform_double(&localState);
    // result[id] += count;
}

int main()
{

    // Initialize the hiprand on device
    data_cpu = (double *)calloc(total_threads, sizeof(double));
    data_cpu_2 = (double *)calloc(total_threads, sizeof(double));
    hipMalloc(&data_gpu, sizeof(double) * total_threads);
    hipMalloc(&data_gpu_2, sizeof(double) * total_threads);
    hipMemset(data_gpu, 0, sizeof(double) * total_threads);
    hipMemset(data_gpu_2, 0, sizeof(double) * total_threads);

    // hipMalloc((void **)&devMRGStates, total_threads * sizeof(hiprandStateMRG32k3a));
    // hipMalloc((void **)&devPHILOXStates, total_threads * sizeof(hiprandStatePhilox4_32_10_t));
    hipMalloc((void **)&devStates, total_threads * sizeof(hiprandState));

    std::cout << "\n data size = " << sizeof(hiprandState);
    std::cout << "\n data size int = " << sizeof(int );

    /* Generate and use pseudo-random  */
    // init_curand_kernel<<<grid_dim, block_dim>>>(devStates);

    generate_kernel<<<grid_dim, block_dim>>>(devStates, sampleCount, data_gpu, data_gpu_2);

    hipMemcpy(data_cpu, data_gpu, total_threads * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(data_cpu_2, data_gpu_2, total_threads * sizeof(double), hipMemcpyDeviceToHost);

    std::cout << std::endl;
    for (size_t i = 0; i < total_threads; i++)
    {
        std::cout << "  " << data_cpu[i];
        if (i % 10 == 0)
        {
            std::cout << "\n";
        }
    }
    std::cout << std::endl;

    for (size_t i = 0; i < total_threads; i++)
    {
        std::cout << "  " << data_cpu_2[i];
        if (i % 10 == 0)
        {
            std::cout << "\n";
        }
    }

    std::cout << std::endl;

    return 0;
}