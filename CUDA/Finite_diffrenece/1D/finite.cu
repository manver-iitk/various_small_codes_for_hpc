#include "hip/hip_runtime.h"
#include "Finite.cuh"

dim3 grid;
dim3 block;

template <typename T>
__global__ void finite_difference(T *input, T *output)
{
    long long int index = threadIdx.x + (blockIdx.x * gridDim.x);

    if (index >= N_gpu)
        return;

    T Front_data;
    T Back_data;
    int spacing;

    if (index == 0)
    {
        // Do the forward difference
        Front_data = input[index + 1];
        Back_data = input[index];
        spacing = 1;
    }
    if (index == (N_gpu - 1))
    {
        // Do the backward difference
        Front_data = input[index];
        Back_data = input[index - 1];
        spacing = 1;
    }
    if ((index > 0) && (index < (N_gpu - 1)))
    {
        Front_data = input[index + 1];
        Back_data = input[index - 1];
        spacing = 2;
    }
    __syncthreads();

    // Diffrenece scheme Formulae
    output[index] = (Front_data - Back_data) / spacing;
}

template <typename T>
void init_arrays_memory()
{
    hipMalloc(&(GPU_input_array<T>), sizeof(T) * N);
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    hipMalloc(&(GPU_output_array<T>), sizeof(T) * N);
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    CPU_input_array<T> = (T *)malloc(sizeof(T) * N);
    CPU_output_array<T> = (T *)malloc(sizeof(T) * N);
}

template <typename T>
void fill_cpu_memory()
{
    for (long long int i = 0; i < N; i++)
    {
        CPU_input_array<T>[i] = i;
    }
}

template <typename T>
void show_data(T *data, int in_out)
{
    if (in_out == 0)
    {
        std::cout << "\n Input data is :  " << std::endl;
    }
    else
    {
        std::cout << "\n Output data is :  " << std::endl;
    }

    for (long long int i = 0; i < N; i++)
    {
        std::cout << data[i] << ",";
    }

    std::cout << "\n\n";
}

template <typename T>
void copy_data_between_devices(int direction)
{
    if (direction == 0)
    {
        hipMemcpy(GPU_input_array<T>, CPU_input_array<T>, sizeof(T) * N, hipMemcpyHostToDevice);
        gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);
    }

    else if (direction == 1)
    {
        hipMemcpy(CPU_output_array<T>, GPU_output_array<T>, sizeof(T) * N, hipMemcpyDeviceToHost);
        gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);
    }
}

int main(int argc, char **argv)
{

    // Setting the grid dimensions
    N = atoi(argv[1]);

    std::cout << "\n N = " << N;

    // Setting the grid and block
    grid = {((N / 256) + 1), 1, 1};
    block = {256, 1, 1};

    // Setting the dimensions to the GPU
    hipMemcpyToSymbol(HIP_SYMBOL(N_gpu), &N, sizeof(long long int));
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    // Allocate the Memoery in CPU
    init_arrays_memory<double>();

    // Initialize the Memoery in CPU
    fill_cpu_memory<double>();

    // Show the data in CPU
    show_data(CPU_input_array<double>, 0);

    // Copy Data from CPU to GPU
    copy_data_between_devices<double>(0);

    // Calling of kernel
    finite_difference<<<grid, block, 0, 0>>>(GPU_input_array<double>, GPU_output_array<double>);

    hipDeviceSynchronize();
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    // Copy Data from GPU to CPU
    copy_data_between_devices<double>(1);

    // Show the data after compuatation
    show_data(CPU_output_array<double>, 1);

    return 0;
}