#include "hip/hip_runtime.h"
#include "Finite.cuh"

template <typename T>
__global__ void finite_difference(T *input, T *output)
{
    long long int i = threadIdx.x + (blockIdx.x * gridDim.x);

    if (i >= (Nx_gpu * Ny_gpu * Nz_gpu))
    {
        return;
    }

    long long int z = i % Nz_gpu;
    long long int y = (i / Nz_gpu) % Ny_gpu;
    long long int x = (i / (Nz_gpu * Ny_gpu)) % Nx_gpu;

    __shared__ T data[256];
    data[threadIdx.x] = input[i];
    __syncthreads();

    if (axis == 0)
    {
        if ((z > 0) && (z < (Nz_gpu - 1)) && (threadIdx.x != 255) && (threadIdx.x != 0))
        {
            output[i] = (data[threadIdx.x + 1] - data[threadIdx.x - 1]) / 2.00;
        }
    }
}

template <typename T>
void init_arrays_memory()
{
    hipMalloc(&(GPU_input_array<T>), sizeof(T) * Grid_dimension);
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    hipMalloc(&(GPU_output_array<T>), sizeof(T) * Grid_dimension);
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    CPU_input_array<T> = (T *)malloc(sizeof(T) * Grid_dimension);
    CPU_output_array<T> = (T *)malloc(sizeof(T) * Grid_dimension);
}

template <typename T>
void fill_cpu_memory()
{
    for (long long int i = 0; i < (Grid_dimension); i++)
    {
        CPU_input_array<T>[i] = i;
    }
}

template <typename T>
void show_data(T *data)
{
    // std::cout << "\n Input data is :  " << std::endl;

    for (long long int i = 0; i < Nx; i++)
    {
        std::cout << "\n\n";
        for (long long int j = 0; j < Ny; j++)
        {
            std::cout << "\n";
            for (long long int k = 0; k < Nz; k++)
            {
                std::cout << "   " << data[(i * Ny * Nz) + (j * Nz) + k];
            }
        }
    }
    std::cout << "\n\n";
}

template <typename T>
void copy_data_between_devices(int direction)
{
    if (direction == 0)
    {
        hipMemcpy(GPU_input_array<T>, CPU_input_array<T>, sizeof(T) * Grid_dimension, hipMemcpyHostToDevice);
        gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);
    }

    else if (direction == 1)
    {
        hipMemcpy(CPU_output_array<T>, GPU_output_array<T>, sizeof(T) * Grid_dimension, hipMemcpyDeviceToHost);
        gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);
    }
}

template <typename T>
void calls_to_program()
{
    init_arrays_memory<T>();
    fill_cpu_memory<T>();
    show_data(CPU_input_array<T>);
    copy_data_between_devices<T>(0);

    // Calling of kernel
    finite_difference<<<{(Nx * Ny * Nz / 256) + 1, 1, 1}, {256, 1, 1}>>>(GPU_input_array<T>, GPU_output_array<T>);

    hipDeviceSynchronize();
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    copy_data_between_devices<T>(1);
    show_data(CPU_output_array<T>);
}

int main(int argc, char **argv)
{

    // Setting the grid dimensions
    Nx = atoi(argv[1]);
    Nx = atoi(argv[2]);
    Nx = atoi(argv[3]);

    std::cout << "\n Nx = " << Nx;
    std::cout << "\n Ny = " << Ny;
    std::cout << "\n Nz = " << Nz;
    Grid_dimension = Nx * Ny * Nz;

    // Setting the dimensions to the GPU
    hipMemcpyToSymbol(HIP_SYMBOL(Nx_gpu), &Nx, sizeof(long long int));
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    hipMemcpyToSymbol(HIP_SYMBOL(Ny_gpu), &Ny, sizeof(long long int));
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    hipMemcpyToSymbol(HIP_SYMBOL(Nz_gpu), &Nz, sizeof(long long int));
    gpuerrcheck_cudaerror(hipGetLastError(), __LINE__ - 1, __FILE__);

    // Start of code
    calls_to_program<double>();

    return 0;
}