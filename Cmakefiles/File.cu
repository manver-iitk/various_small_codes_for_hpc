#include "hip/hip_runtime.h"
#pragma once
#include "header.cuh"

__global__ void test_kernel()
{
    for (size_t i = 0; i < 10; i++)
    {
        printf("\n hello this is id ---> %d", i);
    }
}

int main()
{
    hipDeviceSynchronize();
    test_kernel<<<1, 1, 0, 0>>>();
    hipDeviceSynchronize();

    set_para();

    return 0;
}